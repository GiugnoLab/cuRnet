#include "hip/hip_runtime.h"
/*------------------------------------------------------------------------------
Copyright © 2015 by Nicola Bombieri

H-BF is provided under the terms of The MIT License (MIT):

Permission is hereby granted, free of charge, to any person obtaining a copy of
this software and associated documentation files (the "Software"), to deal in
the Software without restriction, including without limitation the rights to
use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
the Software, and to permit persons to whom the Software is furnished to do so,
subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
------------------------------------------------------------------------------*/
/**
 * @author Federico Busato
 * Univerity of Verona, Dept. of Computer Science
 * federico.busato@univr.it
 */
#include "XLib.hpp"
#include "Host/GraphSSSP.hpp"
#include "Device/HBFGraph.cuh"





//#include "Rcpp.h"
#include <boost/config.hpp>
#include <boost/graph/adjacency_list.hpp>
#include <boost/graph/breadth_first_search.hpp>
#include <boost/graph/visitors.hpp>
#include <boost/graph/graph_utility.hpp>
#include <boost/random.hpp>
#include <boost/random/linear_congruential.hpp>
#include <boost/property_map/property_map.hpp>
#include <boost/graph/erdos_renyi_generator.hpp>
#include <boost/program_options.hpp>
#include <boost/format.hpp>
#include <boost/lexical_cast.hpp>
#include <boost/graph/connected_components.hpp>
#include <boost/graph/dijkstra_shortest_paths.hpp>
#include <boost/graph/prim_minimum_spanning_tree.hpp>
#include <time.h>
#include <stdio.h>
#include <boost/graph/graphviz.hpp>
#include <fstream>
#include <vector>
#include <utility>
#include <string>
#include <math.h>
#include <iomanip>
#include <boost/limits.hpp>
#include <queue>


using namespace boost;
//using namespace Rcpp;
using namespace std;




























bool check_paths(
vector<int>::iterator& first,
vector<int>& terminals,
int *paths
){
int current, pred;
for (std::vector<int>::iterator second = first+1; second!=terminals.end(); ++second) {
	std::vector<int> seen;
	current=*second;
	seen.push_back(current);
	if(paths[current] == -1){
		paths[current] = current;
	}
	pred = paths[current];
	while(pred!=current){
		if( std::find(seen.begin(),seen.end(),pred) != seen.end()){
			for(vector<int>::iterator it = seen.begin(); it!=seen.end(); it++){
				std::cout<<(*it)<<" ";
			}
			std::cout<<pred;
			std::cout<<std::endl;
			return false;
		}

		seen.push_back(pred);
		current=pred;
		pred=paths[current];
		if(pred == -1){
			pred = current;
		}
	}
}
return true;
};





void update_paths(
vector<int>::iterator& first,
vector<int>& terminals,
int *paths,
float *distss,
int outer,
vector <vector<vector<int> > > &perPath,
vector <vector<double> > &Distance
){
int current, pred;
int inner = outer + 1;
for (std::vector<int>::iterator second = first+1; second!=terminals.end(); ++second) {

	std::vector<int> seen;

	current=*second;

	if(distss[current] == std::numeric_limits<float>::max()){
		Distance[outer][inner] = std::numeric_limits<double>::infinity();
		Distance[inner][outer] = std::numeric_limits<double>::infinity();
	}
	else{
		Distance[outer][inner] = distss[current];
		Distance[inner][outer] = distss[current];
	}

	seen.push_back(current);
	if(paths[current] == -1){
		paths[current] = current;
	}
	pred = paths[current];

	while(pred!=current){
		if( std::find(seen.begin(),seen.end(),pred) != seen.end()){
			/*for(vector<int>::iterator it = seen.begin(); it!=seen.end(); it++){
				std::cout<<(*it)<<" ";
			}
			std::cout<<pred;
			std::cout<<std::endl;
			return false;*/
			perPath[outer][inner].push_back(current);
			return;
		}

		perPath[outer][inner].push_back(current);


		seen.push_back(pred);
		current=pred;
		pred=paths[current];
		if(pred == -1){
			pred = current;
		}
	}
	perPath[outer][inner].push_back(current);
	inner++;
}
//return true;
};



void HBF_call_seq_internal(
GraphBasePCSF &g,
vector<int> &terminals,
vector <vector <vector<int> > > &perPath,
vector <vector<double> > &Distance
){
std::cout<<"==================================================================="<<std::endl;
using namespace boost;
using    EdgeIterator = graph_traits<GraphBasePCSF>::edge_iterator;
graph_traits<GraphBasePCSF>::edge_descriptor edge;
using PropMap = boost::property_map<GraphBasePCSF, boost::edge_weight_t>::type;
PropMap edge_weight_map = get(boost::edge_weight, g);
int source;
int dest;
float weight;
EdgeIterator it, end_it;

int V = num_vertices(g);
int E = num_edges(g);

int num_undirected = E * 2;

GraphSSSP graph(V, num_undirected, EdgeType::DIRECTED);
graph.COOSize = num_undirected;

float* weight_array = new float[num_undirected];
int i = 0;
for (boost::tie(it, end_it) = boost::edges(g); it != end_it; ++it) {
	edge = *it;
	source = boost::source(edge, g);
	dest = boost::target(edge, g);
	weight = edge_weight_map[edge];

	graph.COO_Edges[i][0] = source;
	graph.COO_Edges[i][1] = dest;
	weight_array[i] = std::abs(weight);
	graph.COO_Edges[E + i][0] = dest;
	graph.COO_Edges[E + i][1] = source;
	weight_array[E + i] = std::abs(weight);
	i++;
}
graph.ToCSR(weight_array);
std::cout<<"==================================================================="<<std::endl;
HBFGraph devGraph(graph, false);
devGraph.copyToDevice();
//std::vector<Vertex> p = vector<Vertex> (num_vertices(g));
//std::vector<double> d = vector<double> (num_vertices(g));
std::cout<<"==================================================================="<<std::endl;
for(auto& it : terminals){
	std::cout<<it<<" ";
}
std::cout<<std::endl;
std::cout<<"==================================================================="<<std::endl;

int outer = 0;
 for (std::vector<int>::iterator first=terminals.begin(); first!=terminals.end(); ++first){

	weight_t *distss = new weight_t[V];
	int* paths = new int[V];

//    from = vertex(*first, g_adjusted);

	graph.BellmanFord_Queue_init();
	graph.BellmanFord_Queue_reset();
	graph.BellmanFord_Queue(*first);

	weight_t *ww;
	int *pw;
	graph.BellmanFord_Result(ww, pw);

	do{
		std::cout<<"-CU"<<std::endl;
		//devGraph.WorkEfficient_PCSF(*first, distss, paths);
	}while(!check_paths(first, terminals, paths));

	std::vector<Vertex> p = vector<Vertex> (num_vertices(g));
	std::vector<double> d = vector<double> (num_vertices(g));

	Vertex from = vertex(*first, g);
	dijkstra_shortest_paths(g,
				from,
			    	predecessor_map(
					boost::make_iterator_property_map(
						p.begin(),
						get(boost::vertex_index, g)
					)
				)
				.distance_map(
					boost::make_iterator_property_map(
						d.begin(),
						get(boost::vertex_index, g)
					)
				)
	);



	for(int ii=0; ii<V; ii++){
		if( std::abs(ww[ii] - distss[ii]) > 0.0001   &&   std::abs(ww[ii] - distss[ii]) < 1000000.0){
			std::cout<<"S vs P : "<<ii<<" "<<*first<<" : "<<ww[ii] <<" != "<< distss[ii]<<std::endl;
		}
		if( std::abs(ww[ii] - d[ii]) > 0.0001   && std::abs(ww[ii] - d[ii]) < 1000000.0){
			std::cout<<"S vs M : "<<ii<<" "<<*first<<" : "<<ww[ii] <<" != "<< d[ii]<<std::endl;
		}
		if( std::abs(distss[ii] - d[ii]) > 0.0001   && std::abs(distss[ii] - d[ii]) < 1000000.0){
			std::cout<<"P vs M : "<<ii<<" "<<*first<<" : "<<distss[ii] <<" != "<< d[ii]<<std::endl;
		}
	}

	if(! check_paths(first, terminals, paths)){
		std::cout<<"NOT PATH P"<<std::endl;
	}
	if(! check_paths(first, terminals, pw)){
		std::cout<<"NOT PATH S"<<std::endl;
	}

	update_paths(first, terminals, paths,distss,outer,perPath,Distance);
	//update_paths(first, terminals, pw,ww,outer,perPath,Distance);
	outer++;
	delete [] distss;
	delete [] paths;
 }

std::cout<<"==================================================================="<<std::endl;

delete[] weight_array;

};





void HBF_call_seq(
GraphBasePCSF &g,
GraphBasePCSF &g_adjusted,
vector<int> &terminals,
vector <vector <vector<int> > > &perPath,
vector <vector<double> > &Distance
){

GraphBasePCSF bgraph;
using namespace boost;
using    EdgeIterator = graph_traits<GraphBasePCSF>::edge_iterator;
graph_traits<GraphBasePCSF>::edge_descriptor edge;
using PropMap = boost::property_map<GraphBasePCSF, boost::edge_weight_t>::type;
PropMap edge_weight_map = get(boost::edge_weight, g_adjusted);
int source;
int dest;
float weight;
EdgeIterator it, end_it;
for (boost::tie(it, end_it) = boost::edges(g_adjusted); it != end_it; ++it) {
	edge = *it;
	source = boost::source(edge, g_adjusted);
	dest = boost::target(edge, g_adjusted);
	weight = std::abs(edge_weight_map[edge]);
	boost::add_edge(source, dest, std::abs(weight), bgraph);
}

HBF_call_seq_internal(bgraph, terminals, perPath, Distance);

};








































typedef std::vector<std::string> CharacterVector;
typedef std::vector<float> NumericVector;



GraphBasePCSF g, g_adjusted; GraphBasePCSF G; GraphBasePCSF G_pruned;
property_map<GraphBasePCSF, edge_weight_t>::type weight_g;
property_map<GraphBasePCSF, edge_weight_t>::type weight_g_adjusted;
property_map<GraphBasePCSF, edge_weight_t>::type weight_G;
property_map<GraphBasePCSF, edge_weight_t>::type weight_G_pruned;



int Root = -1;
static map <string, int> g_map;
static map <string, int> G_map;
static map <string, int> G_pruned_map;

void clear_variables(){
  g.clear();
  g_adjusted.clear();
  G.clear();
  G_pruned.clear();
  g_map.clear();
  G_map.clear();
  G_pruned_map.clear();
}

// Map
int idx_g(string const & id)
{
  map<string, int>::iterator mit = g_map.find(id);
  if (mit == g_map.end())
    return g_map[id] = add_vertex(VertexPropertiesPCSF(id), g);
  return mit->second;
}
int idx_G(string const & id)
{
  map<string, int>::iterator mit = G_map.find(id);
  if (mit == G_map.end())
    return G_map[id] = add_vertex(VertexPropertiesPCSF(id), G);
  return mit->second;
}
int idx_G_pruned(string const & id)
{
  map<string, int>::iterator mit = G_pruned_map.find(id);
  if (mit == G_pruned_map.end())
    return G_pruned_map[id] = add_vertex(VertexPropertiesPCSF(id), G_pruned);
  return mit->second;
}


//================================================================================================

// Reading the input network
void read_input_graph(CharacterVector from, CharacterVector to,  NumericVector cost, CharacterVector prize, NumericVector prize_v)
{
  for(int i=0; i < from.size(); i++){
    add_edge(vertex(idx_g(from[i]), g), vertex(idx_g(to[i]), g), cost[i], g);
  }
  for(int i=0; i<prize.size(); i++){
    g[idx_g(prize[i])].c = prize_v[i];
  }

  // cerr << num_edges(g) << " edges, " << num_vertices(g) << " vertices" << endl;

  g_adjusted = g;
  weight_g_adjusted = get(edge_weight, g_adjusted);
  graph_edge_iterator ei, ei_end; double penalty; Vertex sour, tar;
  for(tie(ei, ei_end) = edges(g_adjusted); ei != ei_end; ++ei){
    sour = source(*ei,g_adjusted); tar = target(*ei,g_adjusted); penalty = 0;
    if(g_adjusted[sour].c < 0 && g_adjusted[tar].c < 0){
      penalty = g_adjusted[sour].c + g_adjusted[tar].c;
    } else if( g_adjusted[sour].c < 0 ){
      penalty = g_adjusted[sour].c;
    } else if( g_adjusted[tar].c < 0 ){
      penalty = g_adjusted[tar].c;
    }
    weight_g_adjusted[*ei] = weight_g_adjusted[*ei] + abs(penalty);
  }
}












// A function to dynamically remove the leaf NodePCSF if its prize smaller than connection cost.
// It is used called within the process_leafs() function.
void clear(vector <NodePCSF> & predecessor, int & current_NodePCSF){
  int NodePCSF=current_NodePCSF;
  for(unsigned int j=0; j<predecessor[NodePCSF].children.size(); j++){
    clear(predecessor, predecessor[NodePCSF].children[j]);
  }
  predecessor[NodePCSF].father=-1;
}

// A function to dynamically remove the leaf NodePCSF if its prize smaller than connection cost.
void process_leafs(vector <NodePCSF> & predecessor, int & current_NodePCSF, Edge &e, bool &found){
  for(unsigned int i=0; i<predecessor[current_NodePCSF].children.size(); i++){
    process_leafs(predecessor, predecessor[current_NodePCSF].children[i], e, found);
  }
  int NodePCSF = current_NodePCSF;
  if(NodePCSF != predecessor[NodePCSF].father){
    boost::tuples::tie(e,found) = edge( vertex(predecessor[NodePCSF].father, G_pruned),vertex(NodePCSF, G_pruned) , G_pruned);
    if(predecessor[NodePCSF].price - weight_G_pruned[e] <= 0) {
      clear(predecessor, NodePCSF);
    }
  }

}

// A function to dynamically sum up the prizes of vertices.
void price_collect(vector <NodePCSF> & predecessor, int & current_NodePCSF, Edge &e, bool &found){
  for(unsigned int i=0; i<predecessor[current_NodePCSF].children.size(); i++){
    price_collect(predecessor, predecessor[current_NodePCSF].children[i], e, found);
  }
  int NodePCSF = current_NodePCSF;
  if(NodePCSF != predecessor[NodePCSF].father){
    boost::tuples::tie(e,found) = edge(vertex(predecessor[NodePCSF].father, G_pruned),vertex(NodePCSF, G_pruned) , G_pruned);
    if(predecessor[NodePCSF].price - weight_G_pruned[e] > 0)
      predecessor[predecessor[NodePCSF].father].price = predecessor[predecessor[NodePCSF].father].price + predecessor[NodePCSF].price - weight_G_pruned[e];
  }
}




void pa_call(
GraphBasePCSF &g,
GraphBasePCSF &g_adjusted,
vector<int> &terminals,
vector <vector <vector<int> > > &perPath,
vector <vector<double> > &Distance
){
	Vertex from; int current, pred, outer = 0, inner;
	std::vector<Vertex> p = vector<Vertex> (num_vertices(g));
	std::vector<double> d = vector<double> (num_vertices(g));
	for (std::vector<int>::iterator first=terminals.begin(); first!=terminals.end(); ++first){
		from = vertex(*first, g_adjusted);


		dijkstra_shortest_paths(g_adjusted,
					from,
				    	predecessor_map(
						boost::make_iterator_property_map(
							p.begin(),
							get(boost::vertex_index, g_adjusted)
						)
					)
					.distance_map(
						boost::make_iterator_property_map(
							d.begin(),
							get(boost::vertex_index, g_adjusted)
						)
					)
		);

		inner = outer + 1;
		for (std::vector<int>::iterator second = first+1; second!=terminals.end(); ++second) {
			current=*second;
			pred=p[current];
			Distance[outer][inner] = d[current];
			Distance[inner][outer] = d[current];
			while(pred!=current){perPath[outer][inner].push_back(current); current=pred; pred=p[current]; }
			perPath[outer][inner].push_back(current);
			inner++;
		}
		outer++;
	 }
}



// After reading the input network information from the input file, the algorithm constructs a
vector< Vertex >
constructG(vector<int> & terminals, int &Root){

  // Distance: all-pairs-shortest-path distance matrix
  // perPath: List of arcs in all-pairs-shortest-path distance matrix
  vector <vector <vector<int> > > perPath;
  vector <vector<double> > Distance;
  perPath.resize (terminals.size());
  Distance.resize (terminals.size());
  for (unsigned int i = 0; i < terminals.size(); ++i) {
    perPath [i].resize(terminals.size());
    Distance [i].resize(terminals.size());
  }


int current;

  // Computing all-pairs-shortest-path distance matrix
/*  Vertex from; int current, pred, outer = 0, inner;
  std::vector<Vertex> p = vector<Vertex> (num_vertices(g));
  std::vector<double> d = vector<double> (num_vertices(g));

	std::cout<<"NOF VERTICES\t"<< num_vertices(g)<<"\t"<<num_vertices(g_adjusted)<<std::endl;



std::cout<<"======================================================"<<std::endl;
for(auto& it : terminals)
	std::cout<< it <<std::endl;

std::cout<<"======================================================"<<std::endl;




  for (std::vector<int>::iterator first=terminals.begin(); first!=terminals.end(); ++first){

    from = vertex(*first, g_adjusted);
std::cout<<"@ "<<(*first)<<std::endl;


    dijkstra_shortest_paths(	g_adjusted,
				from,
                            	predecessor_map(
					boost::make_iterator_property_map(
						p.begin(),
						get(boost::vertex_index, g_adjusted)
					)
				)
				.distance_map(
					boost::make_iterator_property_map(
						d.begin(),
						get(boost::vertex_index, g_adjusted)
					)
				)

	);

    inner = outer + 1;
    for (std::vector<int>::iterator second = first+1; second!=terminals.end(); ++second) {

//	std::cout<<(*second)<<std::endl;

      	current=*second;
	pred=p[current];

//	std::cout<<(*second)<<" "<<p[current]<<" "<<d[current]<<std::endl;

      Distance[outer][inner] = d[current];
      Distance[inner][outer] = d[current];
      while(pred!=current){perPath[outer][inner].push_back(current); current=pred; pred=p[current]; }

	perPath[outer][inner].push_back(current);

      inner++;
    }
    outer++;

std::cout<<"-----------------------------------------------------"<<std::endl;
  }
*/



//HBF_call_seq(g, g_adjusted, terminals, perPath, Distance);
//int current;




std::cout<<"======================================================"<<std::endl;
/*
  vector <vector <vector<int> > > pa_perPath;
  vector <vector<double> > pa_Distance;
  pa_perPath.resize (terminals.size());
  pa_Distance.resize (terminals.size());
  for (unsigned int i = 0; i < terminals.size(); ++i) {
    pa_perPath [i].resize(terminals.size());
    pa_Distance [i].resize(terminals.size());
  }
	pa_call(g, g_adjusted, terminals, pa_perPath, pa_Distance);
	//HBF_call_seq(g, g_adjusted, terminals, pa_perPath, pa_Distance);
*/
std::cout<<"======================================================"<<std::endl;
	//HBF_call(g, g_adjusted, terminals, perPath, Distance);


HBF_call_seq(g, g_adjusted, terminals, perPath, Distance);


//	int current;
std::cout<<"======================================================--"<<std::endl;



/*
std::cout<<"======================================================"<<std::endl;
//for(auto& it : terminals)
for(long pos =0; pos<terminals.size(); pos++){
	std::cout<< pos<<" "<<terminals[pos] <<std::endl;
}

std::cout<<"======================================================"<<std::endl;
if(Distance.size() != pa_Distance.size()){
	std::cout<<"Distance size diverges: "<<Distance.size()<<" "<<pa_Distance.size()<<std::endl;
}
for(long pos = 0; pos < Distance.size(); pos++){
	if( Distance[pos].size() != pa_Distance[pos].size() ){
		std::cout<<"vector size diverges at "<<pos<<" : "<<Distance[pos].size()<<" "<<pa_Distance[pos].size()<<std::endl;
	}
	else{
		for(long ppos = 0; ppos<Distance[pos].size(); ppos++){
			if(Distance[pos][ppos] != pa_Distance[pos][ppos]){
				std::cout<<"distance diff at "<<pos<<" "<<ppos<<" : "<<Distance[pos][ppos] <<" != "<< pa_Distance[pos][ppos]<<std::endl;
			}
		}
	}
}
std::cout<<"======================================================--"<<std::endl;


if(perPath.size() != pa_perPath.size()){
	std::cout<<"perPath size diverges: "<<perPath.size()<<" "<<pa_perPath.size()<<std::endl;
}

for(long pos = 0; pos < perPath.size(); pos++){
	if( perPath[pos].size() != pa_perPath[pos].size() ){
		std::cout<<"path size diverges at "<<pos<<" : "<<perPath[pos].size()<<" "<<pa_perPath[pos].size()<<std::endl;
	}
	else{
		for(long ppos = 0; ppos<perPath[pos].size(); ppos++){
			if(perPath[pos][ppos].size() != pa_perPath[pos][ppos].size()){
				std::cout<<"pathpath size diverges at "<<pos<<" "<<ppos<<" : "<<perPath[pos][ppos].size() <<" != "<< pa_perPath[pos][ppos].size()<<std::endl;
			}
			else{
				for(long pppos = 0; pppos<perPath[pos][ppos].size(); pppos++){


					if(perPath[pos][ppos][pppos] != pa_perPath[pos][ppos][pppos]){
						std::cout<<"path diff at "<<pos<<" "<<ppos<<" "<<pppos<<" : "<<perPath[pos][ppos][pppos] <<" != "<< pa_perPath[pos][ppos][pppos]<<std::endl;
					}

				}
			}
		}
	}
}
*/
std::cout<<"======================================================--"<<std::endl;



  // Heuristic Clustering, given large input network, the algorithm clusters input network into
  // smaller clusters, and solves the MST afterwards
  set<int> V;
  set<int> D;
  unsigned int root_index = -1;
  vector<int> NodePCSF_labels;
  NodePCSF_labels.resize(terminals.size());
  for(unsigned int i=0; i< NodePCSF_labels.size(); i++){
    NodePCSF_labels[i] = 0;}
  for(unsigned int i=0; i<terminals.size(); i++){
    if(terminals[i] == Root){
      root_index = i;}
    else{
      V.insert(i);}
  }
  NodePCSF_labels[root_index] = INT_MAX;
  int clusterID=0; int targ;

  while(!V.empty()){
    clusterID++;  current = *V.begin();
    NodePCSF_labels[current] =clusterID;
    V.erase(current);
    D.clear();
    for (unsigned int i=0; i < terminals.size(); i++) {
      targ = i;
      if(NodePCSF_labels[targ] == 0 && current != targ && i != root_index){
        if(g[terminals[current]].c >= Distance[current][targ] && g[terminals[targ]].c >= Distance[current][targ]){
          if(g[terminals[targ]].c > 0){
            D.insert(targ); V.erase(targ);
          }
          NodePCSF_labels[targ]=clusterID;
        }
      }
    }

    while(!D.empty()){
      current = *D.begin();  D.erase(current);
      for (unsigned int i=0; i < terminals.size(); i++) {
        targ = i;
        if(NodePCSF_labels[targ] == 0 && current != targ && i != root_index){
          if(g[terminals[current]].c >= Distance[current][targ] && g[terminals[targ]].c >= Distance[current][targ]){
            if(g[terminals[targ]].c > 0){ D.insert(targ); V.erase(targ);}
            NodePCSF_labels[targ]=clusterID;
          }
        }
      }
    }

  }

  // Identfying the vertex membership with respect to clusters
  vector<vector <int> > clusters(clusterID+1);
  for(unsigned int i=0; i< NodePCSF_labels.size(); i++){
    for(int j=0; j<=clusterID; j++){
      if(NodePCSF_labels[i] == j){
        clusters[j].push_back(i);
      }
    }
  }

  std::vector<int>::iterator it, itt; int num_clusters=0;
  for(unsigned int i=1; i<clusters.size(); i++){
    if(clusters[i].size() > 1) num_clusters++;
  }

  if(num_clusters == 0){
    //cout<<"There is no tree in construct G ()"<<endl;
    //return 0;
  }


  // Regrouping the singletone and dobletone clusters after clustering
  unsigned int threshold_num = 2; int min_index; double min_distance;
  for(unsigned int i=1; i<clusters.size(); i++){
    if( clusters[i].size() <= threshold_num){
      for (it=clusters[i].begin(); it!=clusters[i].end(); ++it){
        min_index = -1; min_distance = DBL_MAX;
        for(unsigned int j=1; j<clusters.size(); j++){
          if( clusters[j].size() > threshold_num){
            for (itt=clusters[j].begin(); itt!=clusters[j].end(); ++itt){
              if(min_distance > Distance[*it][*itt] -g[terminals[*it]].c - g[terminals[*itt]].c ){
                min_distance = Distance[*it][*itt] -g[terminals[*it]].c - g[terminals[*itt]].c; min_index = j;
              }
            }
          }
        }


        if (min_index != -1){
          clusters[min_index].push_back(*it);
          *it = -1;
        }
      }
    }

  }



  // Construct an artificial graph G, which is composed of all clusters determined
  // from Heuristic Clustering phase

  string str; unsigned int index1=-1, index2=-1;

  for (unsigned int l = 0; l<terminals.size(); l++){
    str=to_string(l);
    index1=idx_G(str);
    G[index1].c = g[terminals[l]].c;
    G[index1].name = g[terminals[l]].name;
  }


  for (unsigned int l = 0; l<terminals.size(); l++){
    if(l != root_index){
      str=to_string(l);
      index1=idx_G(str);
      add_edge(root_index, index1, Distance[root_index][index1], G);
    }
  }

  for(unsigned int i = 1; i < clusters.size(); i++){
    for (it=clusters[i].begin(); it!=clusters[i].end(); ++it){
      if(*it != -1){
        str=to_string(*it); index1=idx_G(str);
        for (itt=it+1; itt!=clusters[i].end(); ++itt){
          if(*itt != -1){
            str=to_string(*itt); index2=idx_G(str);
            add_edge(index1, index2, Distance[*it][*itt], G);
          }
        }
      }
    }
  }



  weight_G = get(edge_weight, G);
  vector < Vertex > spanning_tree_G(num_vertices(G));
  prim_minimum_spanning_tree(G, & spanning_tree_G[0]);


  Edge beg; Vertex sour, tar; double cost;
  Edge e; bool found;

  weight_g = get(edge_weight, g);

  vector<int> path; index1=0; index2=0;
  edge_iterator out_i, out_end; int add=0;

  // Solving the Minimum Spanning Tree on G
  for(unsigned int i = 0; i < spanning_tree_G.size(); ++i ){

    if(spanning_tree_G[i]!=i ){

      if(i> spanning_tree_G[i]) path=perPath[spanning_tree_G[i]][i];
      else path=perPath[i][spanning_tree_G[i]];

      for(unsigned int j=0; j<path.size()-1; j++){

        sour=vertex(path[j], g); tar= vertex(path[j+1], g);
        boost::tuples::tie(beg, found) = edge(sour, tar,g);
        cost=get(weight_g, beg);

        index1=idx_G_pruned(to_string(path[j])); index2=idx_G_pruned(to_string(path[j+1]));
        add=0;
        for (boost::tuples::tie(out_i, out_end) = out_edges(vertex(index1,G_pruned), G_pruned); out_i != out_end; ++out_i) {
          if(target(*out_i, G_pruned)==index2) add++;
        }
        if(!add) add_edge(index1, index2, cost, G_pruned);
      }
  }
  }

  weight_G_pruned = get(edge_weight, G_pruned);

  vector< Vertex >spanning_tree_G_pruned(num_vertices(G_pruned));
  prim_minimum_spanning_tree(G_pruned, &spanning_tree_G_pruned[0]);


  double total1=0;
  for(unsigned int i = 0; i < spanning_tree_G_pruned.size(); ++i ){
    if(spanning_tree_G_pruned[i] != i){
      sour= vertex(i,G_pruned); tar=vertex(spanning_tree_G_pruned[i], G_pruned);
      boost::tuples::tie(beg, found) = edge(sour, tar,G_pruned);
      total1+=get(weight_G_pruned, beg);
    }
  }


  return spanning_tree_G_pruned;

  }


// After obtaining MST tree, the algorithm prunes the leaf NodePCSFs
// which have prizes smaller than connection cost
double dcut(int &Root, vector< Vertex > &spanning_tree_G_pruned,  vector< string > &tree_from,  vector< string > &tree_to,  vector< double > &tree_cost, map < string, double > &tree_terminals){

  weight_G_pruned = get(edge_weight, G_pruned);

  Edge e; bool found;

  Edge beg; Vertex sour, tar; int ancestor=-1; bool select=false;


  int root= -1;
  if(Root == -1){
    double max=0;
    vertex_iterator ei, ef;
    for(tie(ei, ef)= vertices(G_pruned); ei!=ef; ei++){
      if(g[boost::lexical_cast<int>(G_pruned[*ei].name)].c > max){
        root = *ei;
        max = g[boost::lexical_cast<int>(G_pruned[*ei].name)].c;
      }
    }
  } else {root = idx_G_pruned(to_string(Root));}



  select = true;

  bool ancestor_changed=true; unsigned int father, temp; ancestor = root;
  if (select){
    father=spanning_tree_G_pruned[ancestor];
    spanning_tree_G_pruned[ancestor]=ancestor;
    while(ancestor_changed){
      if(spanning_tree_G_pruned[father]==father){
        ancestor_changed=false;
        spanning_tree_G_pruned[father]=ancestor;
      }else{
        temp=spanning_tree_G_pruned[father];
        spanning_tree_G_pruned[father]=ancestor;
        ancestor=father; father=temp;
      }

    }

  }else{
    //cout <<"There is no tree"<<endl;
    return 0.0;
  }



  vector<NodePCSF> predecessor(num_vertices(G_pruned));
  if(select){
    for(unsigned int i = 0; i < spanning_tree_G_pruned.size(); ++i ){
      if(spanning_tree_G_pruned[i]!=i){
        predecessor[i].father=spanning_tree_G_pruned[i];
        predecessor[spanning_tree_G_pruned[i]].children.push_back(i);
      }else{predecessor[i].father=i;}
    }
  }


  for(unsigned int i = 0; i < predecessor.size(); ++i ){
    predecessor[i].size=predecessor[i].children.size();
    predecessor[i].price=g[boost::lexical_cast<int>(G_pruned[i].name)].c;

  }


  price_collect(predecessor, root, e, found);

  process_leafs(predecessor, root, e, found);

  weight_g = get(edge_weight, g);

  // Tree
  for(unsigned int i = 0; i < predecessor.size(); ++i ){
    if(predecessor[i].father != -1 &&  predecessor[i].father != (int) i ){
      sour= vertex(boost::lexical_cast<int>(G_pruned[i].name),g); tar = vertex(boost::lexical_cast<int>(G_pruned[predecessor[i].father].name),g);
      boost::tuples::tie(beg, found) = edge(sour, tar,g);
      tree_from.push_back(g[boost::lexical_cast<int>(G_pruned[i].name)].name);
      tree_to.push_back(g[boost::lexical_cast<int>(G_pruned[predecessor[i].father].name)].name);
      tree_cost.push_back(weight_g[beg]);
      tree_terminals[g[boost::lexical_cast<int>(G_pruned[i].name)].name] = g[boost::lexical_cast<int>(G_pruned[i].name)].c;
      tree_terminals[g[boost::lexical_cast<int>(G_pruned[predecessor[i].father].name)].name] = g[boost::lexical_cast<int>(G_pruned[predecessor[i].father].name)].c;
    }
  }



  double total = 0, lostPrice =0;
  int uncovered_NodePCSFs = 0;
  for(unsigned int i = 0; i < predecessor.size(); ++i ){
    if(predecessor[i].father != -1){
      if(predecessor[i].father != (int) i){
        sour= vertex(i,G_pruned); tar = vertex(predecessor[i].father,G_pruned);
        boost::tuples::tie(beg, found) = edge(sour, tar,G_pruned);
        total+=get(weight_G_pruned, beg);
      }
    }
  }


  // Lsit of NodePCSFs that are outside of final tree
  vector<int> calculatecost(num_vertices(g));
  for(unsigned int i = 0; i < predecessor.size(); ++i ){
    if(predecessor[i].father != -1 && predecessor[i].father != (int) i){
      sour= vertex(boost::lexical_cast<int>(G_pruned[i].name),g); tar = vertex(boost::lexical_cast<int>(G_pruned[predecessor[i].father].name),g);
      calculatecost[sour]=1; calculatecost[tar]=1;
    }
  }

  // Uncovered NodePCSFs
  for(unsigned int i = 0; i < num_vertices(g); ++i ){
    if(calculatecost[i] == 0 && (int) i != root ){
      lostPrice += g[i].c;
      uncovered_NodePCSFs++;
    }
  }

  // The list of NodePCSFs in the final Tree
  for(unsigned int i = 0; i < num_vertices(g); ++i ){
    if(calculatecost[i] == 1){
    }
  }

  // Objective value
  return total + lostPrice;

}





//List
void call_sr(
CharacterVector from,
CharacterVector to,
NumericVector cost,
CharacterVector NodePCSF_names,
NumericVector NodePCSF_prizes)
{
  clear_variables();

  vector <int> terminals;

  read_input_graph(from, to, cost, NodePCSF_names, NodePCSF_prizes);

  Root = idx_g("DUMMY");

  double max_price=0; int max_price_index = -1;
  for(unsigned int i=0; i<num_vertices(g); i++){
    if (g[i].c > max_price){
      max_price = g[i].c;
      max_price_index = i;
    }
  }

  if(Root != -1){
    for(unsigned int i=0; i<num_vertices(g); i++){
      if( (int) i != Root && g[i].c >0){
        terminals.push_back(i);
      }
    }
  }else{
    Root = max_price_index;
    for(unsigned int i=0; i<num_vertices(g); i++){
      if( (int) i != Root && g[i].c >0){
        terminals.push_back(i);
      }
    }
  }

  terminals.push_back(Root);


  if(terminals.size() <=1){
    // There is no tree
    //return 0;
  }


  vector< Vertex > spanning_tree;
  spanning_tree = constructG(terminals, Root);

  vector< string > tree_from;
  vector< string > tree_to;
  vector< double > tree_cost;
  map < string, double > tree_terminals;
  double obj = dcut(Root, spanning_tree, tree_from, tree_to, tree_cost, tree_terminals);
  if (obj == 0.0) return;

  CharacterVector tree_f(tree_from.size());
  CharacterVector tree_t(tree_to.size());
  NumericVector tree_c(tree_cost.size());
  CharacterVector tree_ter(tree_terminals.size());
  NumericVector tree_ter_p(tree_terminals.size());

  for(unsigned int i=0; i<tree_from.size(); i++){
    tree_f[i]=tree_from[i];
    tree_t[i]=tree_to[i];
    tree_c[i]=tree_cost[i];
  }

  int counter = 0;
  for (std::map<string, double>::iterator it=tree_terminals.begin(); it!=tree_terminals.end(); ++it){
    tree_ter[counter] = it->first;
    tree_ter_p[counter] = it->second;
    counter++;
  }

//  List tree = List::create(tree_from, tree_to, tree_cost, tree_ter, tree_ter_p);
 // return tree;
};

























int main(int argc, char** argv) {

CharacterVector from;
CharacterVector to;
NumericVector cost;
CharacterVector NodePCSF_names;
NumericVector NodePCSF_prizes;


    std::ifstream fin(argv[1]);

	int nof;
	int count;
	std::string str;
	float d_value;

    fin >> nof;
	for(int i=0; i<nof; i++){
		fin>>str;
		from.push_back(str);
	}
fin >> nof;
	for(int i=0; i<nof; i++){
		fin>>str;
		to.push_back(str);
	}

fin >> nof;
	for(int i=0; i<nof; i++){
		fin>>d_value;
		cost.push_back(d_value);
	}

fin >> nof;
	for(int i=0; i<nof; i++){
		fin>>str;
		NodePCSF_names.push_back(str);
	}

fin >> nof;
	for(int i=0; i<nof; i++){
		fin>>d_value;
		NodePCSF_prizes.push_back(d_value);
	}

    fin.close();

	std::cout<<"from "<<from.size()<<std::endl;
	std::cout<<"to "<<to.size()<<std::endl;
	std::cout<<"cost "<<cost.size()<<std::endl;
	std::cout<<"NodePCSF_names "<<NodePCSF_names.size()<<std::endl;
	std::cout<<"NodePCSF_prizes "<<NodePCSF_prizes.size()<<std::endl;

call_sr(from,to,cost,NodePCSF_names,NodePCSF_prizes);

};
